#include "hip/hip_runtime.h"
#include "gpu_math.cuh"
#include <iostream>

__global__ void gpu_add_points_test(Point3D a, Point3D b, Point3D *c)
{
    *c = gpu_add_points(a, b);
}

__global__ void gpu_matrix_mul_3x3_test(float *A, float *B, float *result)
{
    gpu_matrix_mul_3x3(A, B, result);
}

__global__ void gpu_mat_vec_mul_test(float *mat, float *vec, float *res)
{
    gpu_mat_vec_mul(mat, vec, res);
    Point3D tmp = gpu_mat_vec_mul(mat, (Point3D){vec[0], vec[1], vec[2]});

    //Here tmp == res

    res[0] = tmp.x;
    res[1] = tmp.y;
    res[2] = tmp.z;
}

__global__ void gpu_cross_vec3_test(float *u, float *v, float *res)
{
    gpu_cross_vec3(u, v, res);
}

__global__ void gpu_plane_coeffs_test(Point3D vec, Point3D p1, Point3D p2, float *coeffs)
{
    gpu_plane_coeffs(vec, p1, p2, coeffs);
}

__global__ void gpu_distance_from_plane_test(Point3D p, float *coeffs, float *dist)
{
    *dist = gpu_distance_from_plane(p, coeffs);
}

__global__ void gpu_dot_point_test(Point3D p1, Point3D p2, float *result)
{
    *result = gpu_dot_point(p1, p2);
}

int main()
{
    //Matrix multiplication test
    float ma[] = {4.5, 2, 3, 1, 5, 7, 2, 1, 4.5};
    float mb[] = {2, 4, 5, 4, 2, 2, 6, 1, 5};

    float *gpu_ma, *gpu_mb, *gpu_resm;
    float resm[9];

    hipMalloc((void**)&gpu_ma, sizeof(ma));
    hipMalloc((void**)&gpu_mb, sizeof(mb));
    hipMalloc((void**)&gpu_resm, sizeof(mb));

    hipMemcpy(gpu_ma, ma, sizeof(ma), hipMemcpyHostToDevice);
    hipMemcpy(gpu_mb, mb, sizeof(mb), hipMemcpyHostToDevice);

    gpu_matrix_mul_3x3_test<<<1, 1>>>(gpu_ma, gpu_mb, gpu_resm);

    //Adding points test
    Point3D a, b, res, *result;
    a.x = 5; a.y = 7; a.z = 10;
    b.x = 15; b.y = 14; b.z = 4;

    hipMalloc((void **)&result, sizeof(Point3D));

    gpu_add_points_test<<<1, 1>>>(a, b, result);

    //Cross product test
    float u[] = {1, 0, sqrtf(3)};
    float v[] = {1, sqrtf(3), 0};
    float resu[3];

    float *gpu_u, *gpu_v, *gpu_resu;

    hipMalloc((void**)&gpu_u, sizeof(u));
    hipMalloc((void**)&gpu_v, sizeof(v));
    hipMalloc((void**)&gpu_resu, sizeof(v));

    hipMemcpy(gpu_u, u, sizeof(u), hipMemcpyHostToDevice);
    hipMemcpy(gpu_v, v, sizeof(v), hipMemcpyHostToDevice);
    
    gpu_cross_vec3_test<<<1, 1>>>(gpu_u, gpu_v, gpu_resu);
    
    //Plane coefficients test
    Point3D direction = (Point3D) {5, 2, -3};
    Point3D p1 = (Point3D) {1, -1, 3}, p2 = (Point3D) {4, 1, -2};
    float *gpu_coeffs;
    float coeffs[4];

    hipMalloc((void **)&gpu_coeffs, sizeof(coeffs));

    gpu_plane_coeffs_test<<<1, 1>>>(direction, p1, p2, gpu_coeffs);

    hipDeviceSynchronize();

    hipMemcpy(&res, result, sizeof(Point3D), hipMemcpyDeviceToHost);
    hipMemcpy(resm, gpu_resm, sizeof(resm), hipMemcpyDeviceToHost);
    hipMemcpy(resu, gpu_resu, sizeof(resu), hipMemcpyDeviceToHost);
    hipMemcpy(coeffs, gpu_coeffs, sizeof(coeffs), hipMemcpyDeviceToHost);

    hipFree(result);
    hipFree(gpu_ma);
    hipFree(gpu_mb);
    hipFree(gpu_resm);
    hipFree(gpu_u);
    hipFree(gpu_v);
    hipFree(gpu_resu);

    std::cout << "Cross product result:\n";
    for(int i = 0; i < 3; ++i)
    {
        std::cout << resu[i] << " ";
    }

    std::cout << "\nPlane coeffs: \n";
    for(int i = 0; i < 4; ++i)
    {
        std::cout << coeffs[i] << " ";
    }

    std::cout << "\nMatrix multiplication result:\n";
    for(int i = 0; i < 9; ++i)
    {
        std::cout << resm[i] << " ";
    }

    //*****************************
    //Second part of test functions
    //*****************************

    //Point distance from plane test
    Point3D point = (Point3D){2, 3, 1};
    float coeffs_distance[4] = {1, -2, 3, -5};
    float *gpu_dist, *gpu_coeffs_distance, dist;

    hipMalloc((void **)&gpu_dist, sizeof(float));
    hipMalloc((void **)&gpu_coeffs_distance, sizeof(coeffs_distance));

    hipMemcpy(gpu_coeffs_distance, coeffs_distance, sizeof(coeffs_distance), hipMemcpyHostToDevice);

    gpu_distance_from_plane_test<<<1, 1>>>(point, gpu_coeffs_distance, gpu_dist);

    //Matrix vector multiplication test

    float mat[] = {1, 8, 3, 4, 5, 2, 3, 9, 10};
    float vec[] = {1, 5, 4};

    float *gpu_mat, *gpu_vec, *gpu_res_vec;
    float res_vec[3];

    hipMalloc((void**)&gpu_mat, sizeof(mat));
    hipMalloc((void**)&gpu_vec, sizeof(vec));
    hipMalloc((void**)&gpu_res_vec, sizeof(vec));

    hipMemcpy(gpu_mat, mat, sizeof(mat), hipMemcpyHostToDevice);
    hipMemcpy(gpu_vec, vec, sizeof(vec), hipMemcpyHostToDevice);

    gpu_mat_vec_mul_test<<<1, 1>>>(gpu_mat, gpu_vec, gpu_res_vec);

    //Dot product test
    Point3D p_dot1 = (Point3D){2, 4.7, 10};
    Point3D p_dot2 = (Point3D){50, 4, 1.14};

    float *gpu_dot_result, dot_result;

    hipMalloc((void**) &gpu_dot_result, sizeof(float));

    gpu_dot_point_test<<<1, 1>>>(p_dot1, p_dot2, gpu_dot_result);
    
    hipDeviceSynchronize();
    hipMemcpy(&dist, gpu_dist, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&res_vec, gpu_res_vec, sizeof(res_vec), hipMemcpyDeviceToHost);
    hipMemcpy(&dot_result, gpu_dot_result, sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(gpu_dist);
    hipFree(gpu_mat);
    hipFree(gpu_vec);
    hipFree(gpu_res_vec);
    hipFree(gpu_dot_result);
    
    std::cout << "\nPoint distance from plane:\n" << dist;

    std::cout << "\nMatrix vector multiplication result:\n";
    for(int i = 0; i < 3; ++i)
    {
        std::cout << res_vec[i] << " ";
    }

    std::cout << "\nDot product result: " << dot_result;

    std::cout << "\n";

    return 0;
}