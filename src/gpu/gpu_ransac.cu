#include "hip/hip_runtime.h"
#include <iostream>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void gpu_rand_ints(int *rand_nums, int max)
{
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState_t state;
    hiprand_init(ind, 0, 0, &state);
    rand_nums[ind] = hiprand(&state) % max;
}

__device__ int gpu_rand_int(int seed, int max)
{
    hiprandState_t state;
    hiprand_init(seed, 0, 0, &state);
    return hiprand(&state) % max;
}

__global__ void count(const Point3D *points, int *num_of_close_points, int *random_point_indices, int num_of_points)
{
    int ind = threadIdx.x + blockIdx.x * blockDim.x;

    
} 

int main()
{
    //Random number generation
    //************************
    const int num_of_threads = 512;
    const int num = 4096 * 10;

    int *gpu_rand, max = 30000;
    int random_numbers[num];

    hipMalloc((void **) &gpu_rand, sizeof(random_numbers));
    
    //gpu_rand_ints<<<num / num_of_threads, num_of_threads>>>(gpu_rand, max);

    hipDeviceSynchronize();

    hipMemcpy(random_numbers, gpu_rand, num * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(gpu_rand);

    //Allocation test
    //***************

    int *numbers, capacity = 400;
    int *gpu_numbers, *gpu_capacity;

    hipMalloc((void **) &gpu_numbers, capacity * sizeof(float));
    hipMalloc((void **) &gpu_capacity, sizeof(float));

    hipMemcpy(gpu_capacity, &capacity, sizeof(int), hipMemcpyHostToDevice);
    allocate<<<1, 1>>>(gpu_numbers, gpu_capacity);

    hipDeviceSynchronize();

    hipMemcpy(&capacity, gpu_capacity, sizeof(int), hipMemcpyDeviceToHost);

    numbers = new int[capacity]; 

    hipMemcpy(numbers, gpu_numbers, capacity * sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0; i < capacity; ++i)
    {
        std::cout << numbers[i] << " ";
    }
    hipFree(gpu_numbers);
    hipFree(gpu_capacity);
    delete[] numbers;
}