
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

__global__ void set(bool *values, int *indices, int num)
{
  int ind = threadIdx.x;
    values[ indices[ind] ] = true;
}

int main()
{
  bool *gpu_values;
  int *gpu_indices;

  bool values[50];
  bool res_values[50];

  for(int i = 0; i < 50; ++i)
  {
    values[i] = false;
  }

  int indices[25];

  for(int i = 0; i < 25; ++i)
  {
    indices[i] = 2 * i;
  }

  hipMalloc(&gpu_values, 50 * sizeof(bool));
  hipMalloc(&gpu_indices, 50 * sizeof(int));

  hipMemcpy(gpu_values, values, 50 * sizeof(bool), hipMemcpyHostToDevice);
  hipMemcpy(gpu_indices, indices, 25 * sizeof(int), hipMemcpyHostToDevice);
  
  set<<<1, 40>>>(gpu_values, gpu_indices, 25);

  int indices2[15];
  for(int i = 0; i < 15; ++i)
  {
    indices2[i] = i;
  }

  hipMemcpy(gpu_indices, indices2, 15 * sizeof(int), hipMemcpyHostToDevice);
  set<<<1, 40>>>(gpu_values, gpu_indices, 15);
  
  hipMemcpy(res_values, gpu_values, 50 * sizeof(bool), hipMemcpyDeviceToHost);
  
  for(int i = 0; i < 50; ++i)
  {
    std::cout << res_values[i] << " ";
  }

  return 0;
}
