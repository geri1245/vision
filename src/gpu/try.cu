#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <cmath>
#include "../util/point_raw.h"

//Cross product for float vector
__device__ void gpu_cross_3_vec(float *u, float *v, float *res)
{
    res[0] = u[1] * v[2] - u[2] * v[1];
    res[1] = u[2] * v[0] - u[0] * v[2];
    res[2] = u[0] * v[1] - u[1] * v[0];
}

//Cross product for vectors represented by Point3D
__device__ void gpu_cross_3_point(Point3D u, Point3D v, Point3D *res)
{
    res->x = u.y * v.z - u.z * v.y;
    res->y = u.z * v.x - u.x * v.z;
    res->z = u.x * v.y - u.y * v.x;
}

__device__ void gpu_matrix_mul_3x3(float *A, float *B, float *result)
{
    for(int j = 0; j < 3; ++j)
    {
        for(int i = 0; i < 3; ++i)
        {
            //A[i, j] = 3 * i + j;
            result[3 * i + j] = 
                A[3 * i]     * B[j]     + 
                A[3 * i + 1] * B[3 + j] + 
                A[3 * i + 2] * B[6 + j];
        }
    }
}

__device__ void normalize(Point3D *p)
{
    Point3D lp = *p;
    
    float sq = sqrtf(lp.x * lp.x + lp.y * lp.y + lp.z * lp.z);
    
    lp.x = lp.x / sq;
    lp.y = lp.y / sq;
    lp.z = lp.z / sq;

    *p = lp;
}

__device__ Point3D gpu_add_points(Point3D a, Point3D b)
{
    return (Point3D) {a.x + b.x, a.y + b.y, a.z + b.z};
}

__device__ Point3D gpu_sub_points(Point3D a, Point3D b)
{
    Point3D d = (Point3D) {a.x - b.x, a.y - b.y, a.z - b.z};
    return d;
}

__device__ float gpu_distance_from_plane(Point3D p, float *coeffs)
{
    //d = abs( A*x1 + B*y1 + C*z1 ) / sqrt(A^2 + B^2 + C^2) 
    return 
        fabs(coeffs[0] * p.x + coeffs[1] * p.y + coeffs[2] * p.z + coeffs[3]) /
        sqrtf(coeffs[0] * coeffs[0] + coeffs[1] * coeffs[1] + coeffs[2] * coeffs[2]);    
} 

//Calculates coefficients for Ax + By + Cz + D = 0 
//From a direction vector of the plane and 2 points on the plane
__device__ void gpu_plane_coeffs(Point3D vec, Point3D p1, Point3D p2, float *coeffs)
{
    Point3D vec2 = gpu_sub_points(p1, p2); //Calculate another direction vector
    
    Point3D normal;
    gpu_cross_3_point(vec, vec2, &normal); //Now we have the normal vector of the plane
    coeffs[0] = normal.x;
    coeffs[1] = normal.y;
    coeffs[2] = normal.z;
    coeffs[3] = -normal.x * p1.x - normal.y * p1.y - normal.z * p1.z;
}

__global__ void gpu_add_points_test(Point3D a, Point3D b, Point3D *c)
{
    *c = gpu_add_points(a, b);
}

__global__ void gpu_matrix_mul_3x3_test(float *A, float *B, float *result)
{
    gpu_matrix_mul_3x3(A, B, result);
}

__global__ void gpu_cross_3_vec_test(float *u, float *v, float *res)
{
    gpu_cross_3_vec(u, v, res);
}

__global__ void gpu_plane_coeffs_test(Point3D vec, Point3D p1, Point3D p2, float *coeffs)
{
    gpu_plane_coeffs(vec, p1, p2, coeffs);
}

__global__ void gpu_distance_from_plane_test(Point3D p, float *coeffs, float *dist)
{
    *dist = gpu_distance_from_plane(p, coeffs);
}

int main()
{
    //Matrix multiplication test
    float ma[] = {4.5, 2, 3, 1, 5, 7, 2, 1, 4.5};
    float mb[] = {2, 4, 5, 4, 2, 2, 6, 1, 5};

    float *gpu_ma, *gpu_mb, *gpu_resm;
    float resm[9];

    hipMalloc((void**)&gpu_ma, sizeof(ma));
    hipMalloc((void**)&gpu_mb, sizeof(mb));
    hipMalloc((void**)&gpu_resm, sizeof(mb));

    hipMemcpy(gpu_ma, ma, sizeof(ma), hipMemcpyHostToDevice);
    hipMemcpy(gpu_mb, mb, sizeof(mb), hipMemcpyHostToDevice);

    gpu_matrix_mul_3x3_test<<<1, 1>>>(gpu_ma, gpu_mb, gpu_resm);

    //Adding points test
    Point3D a, b, res, *result;
    a.x = 5; a.y = 7; a.z = 10;
    b.x = 15; b.y = 14; b.z = 4;

    hipMalloc((void **)&result, sizeof(Point3D));

    gpu_add_points_test<<<1, 1>>>(a, b, result);

    //Cross product test
    float u[] = {1, 0, sqrtf(3)};
    float v[] = {1, sqrtf(3), 0};
    float resu[3];

    float *gpu_u, *gpu_v, *gpu_resu;

    hipMalloc((void**)&gpu_u, sizeof(u));
    hipMalloc((void**)&gpu_v, sizeof(v));
    hipMalloc((void**)&gpu_resu, sizeof(v));

    hipMemcpy(gpu_u, u, sizeof(u), hipMemcpyHostToDevice);
    hipMemcpy(gpu_v, v, sizeof(v), hipMemcpyHostToDevice);
    
    gpu_cross_3_vec_test<<<1, 1>>>(gpu_u, gpu_v, gpu_resu);
    
    //Plane coefficients test
    Point3D direction = (Point3D) {5, 2, -3};
    Point3D p1 = (Point3D) {1, -1, 3}, p2 = (Point3D) {4, 1, -2};
    float *gpu_coeffs;
    float coeffs[4];

    hipMalloc((void **)&gpu_coeffs, sizeof(coeffs));

    gpu_plane_coeffs_test<<<1, 1>>>(direction, p1, p2, gpu_coeffs);

    hipDeviceSynchronize();

    hipMemcpy(&res, result, sizeof(Point3D), hipMemcpyDeviceToHost);
    hipMemcpy(resm, gpu_resm, sizeof(resm), hipMemcpyDeviceToHost);
    hipMemcpy(resu, gpu_resu, sizeof(resu), hipMemcpyDeviceToHost);
    hipMemcpy(coeffs, gpu_coeffs, sizeof(coeffs), hipMemcpyDeviceToHost);

    hipFree(result);
    hipFree(gpu_ma);
    hipFree(gpu_mb);
    hipFree(gpu_resm);
    hipFree(gpu_u);
    hipFree(gpu_v);
    hipFree(gpu_resu);

    std::cout << "Cross product result:\n";
    for(int i = 0; i < 3; ++i)
    {
        std::cout << resu[i] << " ";
    }

    std::cout << "\nPlane coeffs: \n";
    for(int i = 0; i < 4; ++i)
    {
        std::cout << coeffs[i] << " ";
    }

    std::cout << "\nMatrix multiplication result:\n";
    for(int i = 0; i < 9; ++i)
    {
        std::cout << resm[i] << " ";
    }

    //*****************************
    //Second part of test functions
    //*****************************

    //Point distance from plane test
    Point3D point = (Point3D){2, 3, 1};
    float coeffs_distance[4] = {1, -2, 3, -5};
    float *gpu_dist, *gpu_coeffs_distance, dist;

    hipMalloc((void **)&gpu_dist, sizeof(float));
    hipMalloc((void **)&gpu_coeffs_distance, sizeof(coeffs_distance));

    hipMemcpy(gpu_coeffs_distance, coeffs_distance, sizeof(coeffs_distance), hipMemcpyHostToDevice);

    gpu_distance_from_plane_test<<<1, 1>>>(point, gpu_coeffs_distance, gpu_dist);
    
    hipDeviceSynchronize();
    hipMemcpy(&dist, gpu_dist, sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(gpu_dist);
    
    std::cout << "\nPoint distance from plane:\n" << dist << "\n";

    return 0;
}